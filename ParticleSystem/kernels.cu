#include "hip/hip_runtime.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <>
#include <device_atomic_functions.h>
#include "hip/hip_math_constants.h"
#include "math_functions.h"
#include "stdio.h"

#define THREADS_PER_BLOCK_DIM 256
#define UNIFORM_GRID_MIN 0.0f
#define UNIFORM_GRID_MAX 4.0f
#define PARTICLES_PER_CELL 32
#define CELL_SIZE 0.2f
#define SMOOTHING_KERNEL_RADIUS 0.2f
#define GAS_CONSTANT 0.1
#define RHO_0 1000
#define EPSILON 0.005
#define MU 0.01f

bool first = true;
hipGraphicsResource* vbo_resource;
int* uniformGrid;
int* particlesInCell;
int gridSize;
float gridWidth;
int cellsPerDim;
float* density;
float* pressure;
float3* velocity;

__device__ float smoothing_kernel(float r_sqr, float d)
{
	if (r_sqr < 0 || r_sqr > d*d) return 0;
	float coeff = 315.0f / (64.0f * HIP_PI * pow(d, 9.0f));
	float dist = d*d - r_sqr;
	return coeff * (dist * dist * dist);
}

__device__ float smoothing_kernel_grad(float r_sqr, float d)
{
	if (r_sqr < 0 || r_sqr > d*d) return 0;
	float coeff = -945.0f / (32.0f * HIP_PI * pow(d, 9.0f));
	float dist = d*d - r_sqr;
	return coeff * dist * dist;
}

__device__ float smoothing_kernel_laplacian(float r_sqr, float d)
{
	if (r_sqr < 0 || r_sqr > d*d) return 0;
	float coeff = -945.0f / (32.0f * HIP_PI * pow(d, 9.0f));
	float dist = d*d - r_sqr;
	float weight = 3.0f*d*d - 7.0f*r_sqr;
	return coeff * dist * weight;
}

__device__ float viscosity_kernel_laplacian(float r_len, float d)
{
	float coeff = 45.0f / (HIP_PI * pow(d, 6.0f));
	float dist = d - r_len;
	return coeff*dist;
}
__device__ float3 pressure_kernel_grad(float3 r, float r_len, float d)
{
	float dist = d - r_len;
	if (r_len == 0) return make_float3(0,0,0);
	float coeff = -45.0f / (HIP_PI * pow(d, 6.0f));
	float3 grad = make_float3(r.x*(1.0f / r_len)*(dist*dist), r.y*(1.0f / r_len)*(dist*dist), r.z*(1.0f / r_len)*(dist*dist));
	return make_float3(coeff*grad.x, coeff*grad.y, coeff*grad.z);
}

__global__ void setInitialVelocity_kernel(float3* velocity, int numberParticles)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < numberParticles) {
		velocity[i].x = 0.0f;
		velocity[i].y = 0.0f;
		velocity[i].z = 0.0f;
	}
}

__global__ void calculateDensity_kernel(float* density, float* pressure, const float3* position, int* grid, int* particlesInCell, int numberParticles, int cellsPerDim)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < numberParticles) {
		float x = position[i].x;
		float y = position[i].y;
		float z = position[i].z;

		int cellX = floor(x / CELL_SIZE);
		int cellY = floor(y / CELL_SIZE);
		int cellZ = floor(z / CELL_SIZE);

		float rho = 0;
		for (int j = -1; j <= 1; ++j) {
			for (int k = -1; k <= 1; ++k) {
				for (int l = -1; l <= 1; ++l) {
					int cellXX = cellX + j;
					int cellYY = cellY + k;
					int cellZZ = cellZ + l;
					if (cellXX >= 0 && cellXX < cellsPerDim && 
						cellYY >= 0 && cellYY < cellsPerDim && 
						cellZZ >= 0 && cellZZ < cellsPerDim) {
						int index = cellZZ*cellsPerDim*cellsPerDim*PARTICLES_PER_CELL +
							cellYY*cellsPerDim*PARTICLES_PER_CELL
							+ cellXX*PARTICLES_PER_CELL;
						int cellIndex = cellZZ*cellsPerDim*cellsPerDim + cellYY*cellsPerDim + cellXX;
						for (int particle = 0; particle < particlesInCell[cellIndex]; ++particle) {
							int particleIndex = grid[index + particle];
							if (particleIndex == i) continue;
							float xx = position[particleIndex].x;
							float yy = position[particleIndex].y;
							float zz = position[particleIndex].z;
							float r_sqr = (x - xx)*(x - xx) + (y - yy)*(y - yy) + (z - zz)*(z - zz);
							rho += smoothing_kernel(r_sqr, SMOOTHING_KERNEL_RADIUS);
						}//end of for loop
					}
				}
			}
		}
		density[i] = rho;
		pressure[i] = (pow(rho, 7)-RHO_0)*GAS_CONSTANT;
	}
}

__global__ void integrate_kernel(float3* position, float* density, float* pressure, float3* velocity, int* grid, int* particlesInCell, int numberParticles, int cellsPerDim, float dt)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < numberParticles) {
		float x = position[i].x;
		float y = position[i].y;
		float z = position[i].z;
		float vx = velocity[i].x;
		float vy = velocity[i].y;
		float vz = velocity[i].z;
		int cellX = floor(x / CELL_SIZE);
		int cellY = floor(y / CELL_SIZE);
		int cellZ = floor(z / CELL_SIZE);
		
		float3 f_pressure;
		f_pressure.x = 0;
		f_pressure.y = 0;
		f_pressure.z = 0;
		float3 f_viscosity;
		f_viscosity.x = 0;
		f_viscosity.y = 0;
		f_viscosity.z = 0;
		float3 f_external;
		f_external.x = 0;
		f_external.y = -9.81;
		f_external.z = 0;
		float curr_p = pressure[i];
		
		for (int j = -1; j <= 1; ++j) {
			for (int k = -1; k <= 1; ++k) {
				for (int l = -1; l <= 1; ++l) {
					int cellXX = cellX + j;
					int cellYY = cellY + k;
					int cellZZ = cellZ + l;
					if (cellXX >= 0 && cellXX < cellsPerDim &&
						cellYY >= 0 && cellYY < cellsPerDim &&
						cellZZ >= 0 && cellZZ < cellsPerDim) {
						int index = cellZZ*cellsPerDim*cellsPerDim*PARTICLES_PER_CELL +
							cellYY*cellsPerDim*PARTICLES_PER_CELL
							+ cellXX*PARTICLES_PER_CELL;
						int cellIndex = cellZZ*cellsPerDim*cellsPerDim + cellYY*cellsPerDim + cellXX;
						for (int particle = 0; particle < particlesInCell[cellIndex]; ++particle) {
							int particleIndex = grid[index + particle];
							if (particleIndex == i) continue;
							float xx = position[particleIndex].x;
							float yy = position[particleIndex].y;
							float zz = position[particleIndex].z;
							//calculate f_pressure = -nabla*p(x_i)
							float3 r = make_float3(xx - x, yy - y, zz - z);
							float distance_sqr = (x - xx)*(x - xx) + (y - yy)*(y - yy) + (z - zz)*(z - zz);
							float p = pressure[particleIndex];
							float rho = density[particleIndex];
							//float kernel = smoothing_kernel_grad(distance_sqr, SMOOTHING_KERNEL_RADIUS);
							float3 grad = pressure_kernel_grad(r, sqrt(distance_sqr), SMOOTHING_KERNEL_RADIUS);
							float coeff = ((p + curr_p) / (2 * rho));
							if (p + curr_p == 0 || rho == 0) coeff = 0;
							f_pressure.x -= coeff * grad.x;
							f_pressure.y -= coeff * grad.y;
							f_pressure.z -= coeff * grad.z;
							//calculate f_viscosity = mu*laplace(v(x_i))
							float v_x = velocity[particleIndex].x;
							float v_y = velocity[particleIndex].y;
							float v_z = velocity[particleIndex].z;
							float laplacian = viscosity_kernel_laplacian(sqrt(distance_sqr), SMOOTHING_KERNEL_RADIUS);
							f_viscosity.x += rho==0 ? 0 : ((v_x - vx) / rho * laplacian);
							f_viscosity.y += rho==0 ? 0 : ((v_y - vy) / rho * laplacian);
							f_viscosity.z += rho==0 ? 0 : ((v_z - vz) / rho * laplacian);
						}//end of for loop
					}
				}
			}
		}

		float mu = MU;
		f_viscosity.x *= mu;
		f_viscosity.y *= mu;
		f_viscosity.z *= mu;
		__syncthreads();
		float3 F;
		F.x = f_pressure.x + f_viscosity.x + f_external.x;
		F.y = f_pressure.y + f_viscosity.y + f_external.y;
		F.z = f_pressure.z + f_viscosity.z + f_external.z;
		velocity[i].x = vx + (dt*F.x);
		velocity[i].y = vy + (dt*F.y);
		velocity[i].z = vz + (dt*F.z);

		/*if (position[i].x + dt*velocity[i].x > UNIFORM_GRID_MAX || position[i].x + dt*velocity[i].x < UNIFORM_GRID_MIN) {
			velocity[i].x = 0;
		}
		if (position[i].y + dt*velocity[i].y > UNIFORM_GRID_MAX || position[i].y + dt*velocity[i].y < UNIFORM_GRID_MIN) {
			velocity[i].y = 0;
		}
		if (position[i].z + dt*velocity[i].z > UNIFORM_GRID_MAX || position[i].z + dt*velocity[i].z < UNIFORM_GRID_MIN) {
			velocity[i].z = 0;
		}*/


		position[i].x = position[i].x + dt*velocity[i].x;
		position[i].y = position[i].y + dt*velocity[i].y;
		position[i].z = position[i].z + dt*velocity[i].z;

		if (position[i].x < UNIFORM_GRID_MIN + EPSILON) {
			velocity[i].x *= -0.5f;;
			position[i].x = UNIFORM_GRID_MIN + EPSILON;
		}
		if (position[i].x > UNIFORM_GRID_MAX - EPSILON) {
			velocity[i].x *= -0.5f;
			position[i].x = UNIFORM_GRID_MAX - EPSILON;
		}
		if (position[i].y < UNIFORM_GRID_MIN + EPSILON) {
			velocity[i].y *= -0.5f;
			position[i].y = UNIFORM_GRID_MIN + EPSILON;
		}
		if (position[i].y > UNIFORM_GRID_MAX - EPSILON) {
			velocity[i].y *= -0.5f;
			position[i].y = UNIFORM_GRID_MAX - EPSILON;
		}
		if (position[i].z < UNIFORM_GRID_MIN + EPSILON) {
			velocity[i].z *= -0.5f;
			position[i].z = UNIFORM_GRID_MIN + EPSILON;
		}
		if (position[i].z > UNIFORM_GRID_MAX - EPSILON) {
			velocity[i].z *= -0.5f;
			position[i].z = UNIFORM_GRID_MAX - EPSILON;
		}
	}
}

__global__ void clearGrid_kernel(int* grid, int* particlesInCell, int gridSize)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	if (x < gridSize) {
		particlesInCell[x] = 0;
		for(int i = 0; i < PARTICLES_PER_CELL; ++i)
			grid[x*PARTICLES_PER_CELL +i] = -1;
	}
}

__global__ void updateGrid_kernel(int* grid, int* particlesInCell, const float3* position, size_t numParticles, size_t cellsPerDim)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;	//index of particle
	if (i < numParticles) {
		float x = position[i].x;
		float y = position[i].y;
		float z = position[i].z;

		int cellX = floor(x / CELL_SIZE);
		int cellY = floor(y / CELL_SIZE);
		int cellZ = floor(z / CELL_SIZE);
		int index = cellZ*cellsPerDim*cellsPerDim*PARTICLES_PER_CELL +
			cellY*cellsPerDim*PARTICLES_PER_CELL +
			cellX*PARTICLES_PER_CELL;
		int cellIndex = cellZ*cellsPerDim*cellsPerDim + cellY*cellsPerDim + cellX;
		int offset = atomicAdd(&(particlesInCell[cellIndex]), 1);
		if(offset < PARTICLES_PER_CELL)
			grid[index+offset] = i;
		//printf("Thread:%i, Offset:%i\n", i, offset);
	}

}

// just for calculation of blocks in grid
int iDivUp(int x, int y)
{
	return x / y + (x % y != 0);
}

void integrate(GLuint vbo, size_t numParticles, float dt)
{
	if (first) {
		hipGraphicsGLRegisterBuffer(&vbo_resource, vbo, hipGraphicsRegisterFlagsNone);
		gridWidth = UNIFORM_GRID_MAX - UNIFORM_GRID_MIN;
		gridSize = (gridWidth/ CELL_SIZE)*(gridWidth / CELL_SIZE)*(gridWidth / CELL_SIZE);
		cellsPerDim = gridWidth / CELL_SIZE;
		hipMalloc(&uniformGrid, gridSize * sizeof(int)*PARTICLES_PER_CELL);
		hipMalloc(&particlesInCell, gridSize * sizeof(int));
		hipMalloc(&density, numParticles * sizeof(float));
		hipMalloc(&pressure, numParticles * sizeof(float));
		
		hipMalloc(&velocity, numParticles * sizeof(float3));
		dim3 threads_in_block(THREADS_PER_BLOCK_DIM, 1, 1);
		dim3 blocks_in_grid = dim3(iDivUp(numParticles, threads_in_block.x), 1, 1);
		setInitialVelocity_kernel << <blocks_in_grid, threads_in_block >> > (velocity, numParticles);
		first = false;
	}
	float3* positions;
	hipGraphicsMapResources(1, &vbo_resource, 0);

	size_t N;	//Number of bytes in the position buffer

	hipGraphicsResourceGetMappedPointer((void**)&positions, &N, vbo_resource);

	dim3 threads_in_block(min(numParticles,THREADS_PER_BLOCK_DIM), 1, 1);

	dim3 blocks_in_grid(iDivUp(gridSize, threads_in_block.x), 1, 1);
	clearGrid_kernel<<<blocks_in_grid, threads_in_block>>>(uniformGrid, particlesInCell, gridSize);


	blocks_in_grid = dim3(iDivUp(numParticles, threads_in_block.x), 1, 1);

	updateGrid_kernel << <blocks_in_grid, threads_in_block >> > (uniformGrid, particlesInCell, positions, numParticles, cellsPerDim);
	calculateDensity_kernel << <blocks_in_grid, threads_in_block >> >(density, pressure, positions, uniformGrid, particlesInCell, numParticles, cellsPerDim);
	integrate_kernel<<<blocks_in_grid, threads_in_block>>>(positions,density, pressure, velocity, uniformGrid, particlesInCell, numParticles, cellsPerDim, 0.01f);
	hipGraphicsUnmapResources(1, &vbo_resource, 0);
}


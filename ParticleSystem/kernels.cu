#include "hip/hip_runtime.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <>
#include <device_atomic_functions.h>
#include "hip/hip_math_constants.h"
#include "math_functions.h"
#include "stdio.h"

#define THREADS_PER_BLOCK_DIM 256
#define UNIFORM_GRID_MIN 0.0f
#define UNIFORM_GRID_MAX 4.0f
#define PARTICLE_SIZE 0.2f
#define ELASTICITY 0.05f
#define INERTIA 0.2f
#define GROUND_ELASTICITY 0.5f

bool first = true;
hipGraphicsResource* vbo_resource;
int* uniformGrid;
int gridSize;
float gridWidth;
int cellsPerDim;
float3* velocity;

__device__ int3 calculateCell(const float3& position)
{
	float x = position.x;
	float y = position.y;
	float z = position.z;

	int cellX = floor(x / PARTICLE_SIZE);
	int cellY = floor(y / PARTICLE_SIZE);
	int cellZ = floor(z / PARTICLE_SIZE);

	return make_int3(cellX, cellY, cellZ);
}

__global__ void setInitialVelocity_kernel(float3* velocity, int numberParticles)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < numberParticles) {
		velocity[i].x = 0.0f;
		velocity[i].y = 0.0f;
		velocity[i].z = 0.0f;
	}
}

__global__ void clearGrid_kernel(int* grid, int gridSize)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	if (x < gridSize) {
		grid[x] = -1;
	}
}

__global__ void updateGrid_kernel(int* grid, const float3* position, size_t numParticles, size_t cellsPerDim)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;	//index of particle
	if (i < numParticles) {
		int3 cell = calculateCell(position[i]);
		int index = cell.z*cellsPerDim*cellsPerDim +
			cell.y*cellsPerDim +
			cell.x;
		grid[index] = i;
		//printf("Thread:%i, Offset:%i\n", i, offset);
	}
}

__device__ float3 difference(float3 vector1, float3 vector2)
{
	float x = vector1.x - vector2.x;
	float y = vector1.y - vector2.y;
	float z = vector1.z - vector2.z;
	return make_float3(x, y, z);
}

__device__ float3 add(float3 vector1, float3 vector2)
{
	float x = vector1.x + vector2.x;
	float y = vector1.y + vector2.y;
	float z = vector1.z + vector2.z;
	return make_float3(x, y, z);
}

__device__ float3 mult(float scalar, float3 vector)
{
	float x = scalar*vector.x;
	float y = scalar*vector.y;
	float z = scalar*vector.z;
	return make_float3(x, y, z);
}

__device__ float dot(float3 vector1, float3 vector2)
{
	return vector1.x*vector2.x + vector1.y*vector2.y + vector1.z*vector2.z;
}

__device__ float norm(float3 vector)
{
	return sqrt(dot(vector, vector));
}

__device__ float3 cross(float3 vector1, float3 vector2)
{
	float x = vector1.y * vector2.z - vector1.z*vector2.y;
	float y = vector1.z * vector2.x - vector1.x*vector2.z;
	float z = vector1.x * vector2.y - vector1.y*vector2.x;
	return make_float3(x, y, z);
}

//returns velocity of the FIRST particle after collision
__device__ float3 collide(float3 position1, float3 position2, float3 velocity1, float3 velocity2)
{
	//transform into a coordinate system in which particle2 doesn't move
	float3 velTransformed = difference(velocity2, velocity1);
	//project the velocity onto the relative position
	float3 r = difference(position2, position1);
	float len = norm(r);
	float3 r_normed = make_float3(r.x / len, r.y / len, r.z / len);
	float coeff = dot(r_normed, velTransformed);
	float3 velLinear = make_float3(coeff*r_normed.x, coeff*r_normed.y, coeff*r_normed.z);
	float3 velPerpendicular = make_float3(velTransformed.x - velLinear.x, velTransformed.y - velLinear.y, velTransformed.z - velLinear.z);
	float3 newVel = make_float3(ELASTICITY*velLinear.x + INERTIA*velPerpendicular.x - (2 * PARTICLE_SIZE)*r_normed.x,
								ELASTICITY*velLinear.y + INERTIA*velPerpendicular.x - (2 * PARTICLE_SIZE)*r_normed.y,
								ELASTICITY*velLinear.z + INERTIA*velPerpendicular.z - (2 * PARTICLE_SIZE)*r_normed.z
								);
	return newVel;
}

__global__ void collideWithNeighbors_kernel(float3* position, float3* velocity, int* grid, int cellsPerDim, int numParticles)
{
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread < numParticles) {
		float3 pos = position[thread];
		float3 vel = velocity[thread];
		float3 collisionVel = make_float3(0.0f, 0.0f, 0.0f);
		int3 cell = calculateCell(pos);
		for (int xOffset = -1; xOffset < 2; xOffset += 2) {
			int cellX = cell.x + xOffset;
			if (cellX < 0 || cellX >= cellsPerDim) continue;
			for (int yOffset = -1; yOffset < 2; yOffset += 2) {
				int cellY = cell.x + yOffset;
				if (cellY < 0 || cellY >= cellsPerDim) continue;
				for (int zOffset = -1; zOffset < 2; zOffset += 2) {
					int cellZ = cell.z + zOffset;
					if (cellZ < 0 || cellZ >= cellsPerDim) continue;

					int neighboringParticle = grid[cellZ*cellsPerDim*cellsPerDim + cellY*cellsPerDim + cellX];
					if (neighboringParticle == -1) continue;
					float3 neighborPos = position[neighboringParticle];
					float3 neighborVel = velocity[neighboringParticle];
					float3 r = difference(neighborPos, pos);
					float distSquared = dot(r, r);
					if (distSquared < (2 * PARTICLE_SIZE)*(2 * PARTICLE_SIZE)) {	//collision detected
						
						collisionVel = collide(pos, neighborPos, vel, neighborVel);

					}
				}
			}
		}
		__syncthreads();

		velocity[thread] = add(vel, collisionVel);
	}
}

__global__ void step(float3* position, float3* velocity, int numParticles, float dt)
{
	int thread = blockIdx.x*blockDim.x + threadIdx.x;
	if (thread < numParticles) {
		float3 pos = position[thread];
		float3 vel = velocity[thread];
		pos = add(pos, mult(dt, vel));

		//apply gravity
		vel.y -= 0.005f;
		
		//Collision with boundary
		if (pos.x - PARTICLE_SIZE < UNIFORM_GRID_MIN) {
			pos.x = UNIFORM_GRID_MIN + PARTICLE_SIZE;
			vel.x *= -GROUND_ELASTICITY;
		}
		if (pos.x + PARTICLE_SIZE > UNIFORM_GRID_MAX) {
			pos.x = UNIFORM_GRID_MAX - PARTICLE_SIZE;
			vel.x *= -GROUND_ELASTICITY;
		}

		if (pos.y - PARTICLE_SIZE < UNIFORM_GRID_MIN) {
			pos.y = UNIFORM_GRID_MIN + PARTICLE_SIZE;
			vel.y *= -GROUND_ELASTICITY;
		}
		if (pos.y + PARTICLE_SIZE > UNIFORM_GRID_MAX) {
			pos.y = UNIFORM_GRID_MAX - PARTICLE_SIZE;
			vel.y *= -GROUND_ELASTICITY;
		}

		if (pos.z - PARTICLE_SIZE < UNIFORM_GRID_MIN) {
			pos.z = UNIFORM_GRID_MIN + PARTICLE_SIZE;
			vel.z *= -GROUND_ELASTICITY;
		}
		if (pos.z + PARTICLE_SIZE > UNIFORM_GRID_MAX) {
			pos.z = UNIFORM_GRID_MAX - PARTICLE_SIZE;
			vel.z *= -GROUND_ELASTICITY;
		}

		__syncthreads();

		position[thread] = pos;
		velocity[thread] = vel;
	}
}

// just for calculation of blocks in grid
int iDivUp(int x, int y)
{
	return x / y + (x % y != 0);
}

void simulate(GLuint vbo, size_t numParticles, float dt)
{
	if (first) {
		hipGraphicsGLRegisterBuffer(&vbo_resource, vbo, hipGraphicsRegisterFlagsNone);
		gridWidth = UNIFORM_GRID_MAX - UNIFORM_GRID_MIN;
		gridSize = (gridWidth/ PARTICLE_SIZE)*(gridWidth / PARTICLE_SIZE)*(gridWidth / PARTICLE_SIZE);
		cellsPerDim = gridWidth / PARTICLE_SIZE;
		hipMalloc(&uniformGrid, gridSize * sizeof(int));
		
		hipMalloc(&velocity, numParticles * sizeof(float3));
		dim3 threads_in_block(THREADS_PER_BLOCK_DIM, 1, 1);
		dim3 blocks_in_grid = dim3(iDivUp(numParticles, threads_in_block.x), 1, 1);
		setInitialVelocity_kernel << <blocks_in_grid, threads_in_block >> > (velocity, numParticles);
		first = false;
	}
	float3* positions;
	hipGraphicsMapResources(1, &vbo_resource, 0);

	size_t N;	//Number of bytes in the position buffer

	hipGraphicsResourceGetMappedPointer((void**)&positions, &N, vbo_resource);

	dim3 threads_in_block(min(numParticles,THREADS_PER_BLOCK_DIM), 1, 1);

	dim3 blocks_in_grid(iDivUp(gridSize, threads_in_block.x), 1, 1);
	clearGrid_kernel<<<blocks_in_grid, threads_in_block>>>(uniformGrid, gridSize);

	blocks_in_grid = dim3(iDivUp(numParticles, threads_in_block.x), 1, 1);

	step << <blocks_in_grid, threads_in_block >> > (positions, velocity, numParticles, dt);
	updateGrid_kernel << <blocks_in_grid, threads_in_block >> > (uniformGrid, positions, numParticles, cellsPerDim);
	collideWithNeighbors_kernel << <blocks_in_grid, threads_in_block >> > (positions, velocity, uniformGrid, cellsPerDim, numParticles);
	hipGraphicsUnmapResources(1, &vbo_resource, 0);
}

